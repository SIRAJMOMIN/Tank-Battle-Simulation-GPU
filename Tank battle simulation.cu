

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

//*******************************************

// Write down the kernels here

__global__ void health(int *gh,int H,int *alive,int *gs)
{
    unsigned id=threadIdx.x;
    gh[id]=H;
    alive[id]=1;
    gs[id]=0;
}

__global__ void dkernel(int T,int *gh,int *gs,int *gx,int *gy,int *alive,int *count)
{
    unsigned id=threadIdx.x;
    int currentround=1;
    while(count[0]>1)
    {
       if(currentround%T==0)
       {
        currentround++;
        __syncthreads();
        continue;
       }
       int target=(id+currentround)%T;
        int m1=(gx[target]-gx[id]);
        int m2=(gy[target]-gy[id]);
        int distance=1e9;
        int shootindex=-1;
       if(alive[id]==1)
       {

        for(int i=0;i<T;i++)
        {   int a1=gx[i]-gx[id];
            int a2=gy[i]-gy[id];
            int dis1=a1;
            if(a1<0)
            dis1=-a1;
            int dis2=a2;
            if(a2<0)
            dis2=-a2;
            int d=dis1+dis2;
            if(i!=id && alive[i]==1 && a1*m2==a2*m1)
            {

                    if((m1<0 && a1<0) || (m2<0 &&  a2<0) || (m1>0 && a1>0) || (m2>0 && a2>0)) // if the point is in the direction of target.
                    {
                        int y=d;
                        if(distance>y)
                        {
                            distance=y;
                            shootindex=i;

                        }

                    }

            }
        }
       }
      if(shootindex!=-1)
      {
        gs[id]++;
        atomicSub(&gh[shootindex],1);
      }
      count[0]=0;

      __syncthreads();
      currentround++;
      if(gh[id]<=0)
      {
          alive[id]=0;
      }
      if(gh[id]>0){
        atomicAdd(&count[0],1);

      }
      __syncthreads();



    }

}


//***********************************************


int main(int argc,char **argv)
{
    // Variable declarations
    int M,N,T,H,*xcoord,*ycoord,*score;


    FILE *inputfilepointer;

    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0;
    }

    fscanf( inputfilepointer, "%d", &M );
    fscanf( inputfilepointer, "%d", &N );
    fscanf( inputfilepointer, "%d", &T ); // T is number of Tanks
    fscanf( inputfilepointer, "%d", &H ); // H is the starting Health point of each Tank

    // Allocate memory on CPU
    xcoord=(int*)malloc(T * sizeof (int));  // X coordinate of each tank
    ycoord=(int*)malloc(T * sizeof (int));  // Y coordinate of each tank
    score=(int*)malloc(T * sizeof (int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for(int i=0;i<T;i++)
    {
      fscanf( inputfilepointer, "%d", &xcoord[i] );
      fscanf( inputfilepointer, "%d", &ycoord[i] );
    }


    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************

    int *gh;
    hipMalloc(&gh,T*sizeof(int));
    int *alive;
    hipMalloc(&alive,T*sizeof(int));
    int *gs;
    hipMalloc(&gs,T*sizeof(int));
    health<<<1,T>>>(gh,H,alive,gs);
    int *gx;
    hipMalloc(&gx,T*sizeof(int));
    hipMemcpy(gx,xcoord,T*sizeof(int),hipMemcpyHostToDevice);
    int *gy;
    hipMalloc(&gy,T*sizeof(int));
    hipMemcpy(gy,ycoord,T*sizeof(int),hipMemcpyHostToDevice);
    int count=T;
    int *gpu_count;
    hipMalloc(&gpu_count,1*sizeof(int));
    hipMemcpy(gpu_count,&count,1*sizeof(int),hipMemcpyHostToDevice);
    dkernel<<<1,T>>>(T,gh,gs,gx,gy,alive,gpu_count);
    hipMemcpy(score,gs,T*sizeof(int),hipMemcpyDeviceToHost);

    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end  = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end-start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3];
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    for(int i=0;i<T;i++)
    {
        fprintf( outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename,"w");
    fprintf(outputfilepointer,"%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}